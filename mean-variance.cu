#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1000000

__global__ void calculateMean(int* data, float* mean) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Compute mean
    atomicAdd(mean, data[tid]);

    // Print information
    //printf("Thread %d: Adding %f to mean\n", tid, data[tid]);
}

__global__ void calculateVariance(int* data, float mean, float* variance) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    //printf("The mean for variance is %f\n", mean);
    if (tid < N) {
        // Synchronize threads to ensure accurate mean calculation

        // Compute variance
        float diff = data[tid] - mean;
        atomicAdd(variance, diff * diff);

        // Print information
        //printf("Thread %d: Adding %f to variance\n", tid, diff * diff);
    }
}

int main() {
    // Allocate memory on host
    int* h_data = (int*)malloc(N * sizeof(int));
    float* h_mean = (float*)malloc(sizeof(float));
    float* h_variance = (float*)malloc(sizeof(float));

    // Initialize the array with values 1 to N, in this case 1,000,000
    for (int i = 0; i < N; i++) {
        h_data[i] = i + 1;
        //printf("%d ", h_data[i]);
    }
    //printf("\n");
    int* d_data;
    float * d_mean, * d_variance;
    hipMalloc((void**)&d_data, N * sizeof(int));
    hipMalloc((void**)&d_mean, sizeof(float));
    hipMalloc((void**)&d_variance, sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_data, h_data, N * sizeof(int), hipMemcpyHostToDevice);

    // Set initial values for mean and variance on host
    *h_mean = 0.0f;
    *h_variance = 0.0f;

    // Copy initial values from host to device
    hipMemcpy(d_mean, h_mean, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_variance, h_variance, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel to calculate mean
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    calculateMean <<<gridSize, blockSize >>> (d_data, d_mean);

    // Synchronize and copy mean result from device to host
    hipMemcpy(h_mean, d_mean, sizeof(float), hipMemcpyDeviceToHost);

    // Compute final mean
    *h_mean /= N;

    // Launch kernel to calculate variance
    calculateVariance << <gridSize, blockSize >> > (d_data, *h_mean, d_variance);

    // Synchronize and copy variance result from device to host
    hipMemcpy(h_variance, d_variance, sizeof(float), hipMemcpyDeviceToHost);

    // Compute final variance, specifically the sample variance, if we want population variance we get rid of the -1
    *h_variance /= (N - 1);

    // Display results
    printf("\nMean: %f\n", *h_mean);
    printf("Variance: %f\n", *h_variance);

    // Free allocated memory
    free(h_data);
    free(h_mean);
    free(h_variance);
    hipFree(d_data);
    hipFree(d_mean);
    hipFree(d_variance);

    return 0;
}

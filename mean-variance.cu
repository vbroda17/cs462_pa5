// Vincent Broda
// CS462 Assignment 5, mean and variance calculations with cuda. See read me for more specific explinations of code and ideas.

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N 1000000

// using atomic add to calculate the summation of all the data in the set
__global__ void calculateMean(int* data, float* mean) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    atomicAdd(mean, data[tid]);
}

// doing something simlar, this time the summation that we need is slightly different though
__global__ void calculateVariance(int* data, float mean, float* variance) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // if (tid < N) {     // Originally had this if statement to try to do some type of check, but I think it was just becasuse of my paranorma and i removed it. I did this after my test on the isaac node though 
    float diff = data[tid] - mean;
    atomicAdd(variance, diff * diff);
}

int main() {
    // Allocate memory on host
    int* h_data = (int*)malloc(N * sizeof(int));
    float* h_mean = (float*)malloc(sizeof(float));
    float* h_variance = (float*)malloc(sizeof(float));

    // Memory for the device
    int* d_data;
    float * d_mean, * d_variance;
    hipMalloc((void**)&d_data, N * sizeof(int));
    hipMalloc((void**)&d_mean, sizeof(float));
    hipMalloc((void**)&d_variance, sizeof(float));

    // Initialize the array with values 1 to N, in this case 1,000,000
    for (int i = 0; i < N; i++) h_data[i] = i + 1;

    // Copy data set from host to device
    hipMemcpy(d_data, h_data, N * sizeof(int), hipMemcpyHostToDevice);
    *h_mean = 0.0f;
    *h_variance = 0.0f;

    // Copy initial values from host to device, I dont think this is actually needed but all well, I'm sure it hurts preformance a little but it is not noticable
    hipMemcpy(d_mean, h_mean, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_variance, h_variance, sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel to calculate mean, we will alco be using these block sizes, I'm not sure if theres a better way to decide this or not, but again the preformance is good enough for it to be fine I belive 
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    calculateMean<<<gridSize, blockSize>>>(d_data, d_mean);

    // Synchronize result from device to host
    hipMemcpy(h_mean, d_mean, sizeof(float), hipMemcpyDeviceToHost);

    // Compute final mean, we will do division here for efficiency
    *h_mean /= N;

    // Launch kernel to calculate variance, very similar to mean, but just a bit more complicated becasue it is the variance
    calculateVariance<<<gridSize, blockSize>>>(d_data, *h_mean, d_variance);

    // Synchronize result from device to host
    hipMemcpy(h_variance, d_variance, sizeof(float), hipMemcpyDeviceToHost);

    // Compute final variance, specifically the sample variance, if we want population variance we get rid of the -1
    *h_variance /= (N - 1);

    printf("Mean: %f\n", *h_mean);
    printf("Variance: %f\n", *h_variance);

    // Free allocated memory
    free(h_data);
    free(h_mean);
    free(h_variance);
    hipFree(d_data);
    hipFree(d_mean);
    hipFree(d_variance);

    return 0;
}
